#include "hip/hip_runtime.h"
#include <RayTracerUtilities.cuh>

namespace evo_engine {
extern "C" __constant__ PointCloudScanningLaunchParams pointCloudScanningLaunchParams;

struct PointCloudScanningPerRayData {
  bool hit;
  Random random;
  uint64_t handle;
  HitInfo hit_info;
};

#pragma region Closest hit functions
extern "C" __global__ void __closesthit__PCS_R() {
  const auto &sbtData = *(const SBT *)optixGetSbtDataPointer();
  const float2 triangleBarycentricsInternal = optixGetTriangleBarycentrics();
  const int primitiveId = optixGetPrimitiveIndex();
  const float3 rayDirectionInternal = optixGetWorldRayDirection();
  glm::vec3 rayDirection = glm::vec3(rayDirectionInternal.x, rayDirectionInternal.y, rayDirectionInternal.z);
  auto hit_info = sbtData.GetHitInfo(rayDirection);

  PointCloudScanningPerRayData &prd = *GetRayDataPointer<PointCloudScanningPerRayData>();
  prd.hit = true;
  prd.handle = sbtData.m_handle;
  prd.hit_info = hit_info;
}
extern "C" __global__ void __closesthit__PCS_SS() {
}
#pragma endregion
#pragma region Any hit functions
extern "C" __global__ void __anyhit__PCS_R() {
}
extern "C" __global__ void __anyhit__PCS_SS() {
}
#pragma endregion
#pragma region Miss functions
extern "C" __global__ void __miss__PCS_R() {
  PointCloudScanningPerRayData &prd = *GetRayDataPointer<PointCloudScanningPerRayData>();
  prd.hit = false;
  prd.handle = 0;
}
extern "C" __global__ void __miss__PCS_SS() {
}
#pragma endregion
#pragma region Main ray generation
extern "C" __global__ void __raygen__PCS() {
  unsigned ix = optixGetLaunchIndex().x;
  auto &samples = pointCloudScanningLaunchParams.m_samples[ix];
  auto start = samples.m_start;
  auto direction = samples.m_direction;
  float3 rayOrigin = make_float3(start.x, start.y, start.z);
  float3 rayDirection = make_float3(direction.x, direction.y, direction.z);

  PointCloudScanningPerRayData perRayData;
  perRayData.random.Init(ix, 0);
  perRayData.hit = false;
  perRayData.hit_info = HitInfo();
  uint32_t u0, u1;
  PackRayDataPointer(&perRayData, u0, u1);
  optixTrace(pointCloudScanningLaunchParams.m_traversable, rayOrigin, rayDirection,
             1e-3f,  // tmin
             1e20f,  // tmax
             0.0f,   // rayTime
             static_cast<OptixVisibilityMask>(255),
             OPTIX_RAY_FLAG_DISABLE_ANYHIT,            // OPTIX_RAY_FLAG_NONE,
             static_cast<int>(RayType::Radiance),      // SBT offset
             static_cast<int>(RayType::RayTypeCount),  // SBT stride
             static_cast<int>(RayType::Radiance),      // missSBTIndex
             u0, u1);
  samples.m_handle = perRayData.handle;
  samples.m_hit = perRayData.hit;
  samples.m_hitInfo = perRayData.hit_info;
}
#pragma endregion
}  // namespace evo_engine
